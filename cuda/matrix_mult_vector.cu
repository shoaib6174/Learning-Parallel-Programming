#include "hip/hip_runtime.h"
#include<iostream>
#include "my_utils.h"
#include<cmath>

__global__
void matrix_vec_mult_kernel(float* A_d, float* B_d, float* C_d, int vec_len)
{
    row = blockIdx.x * blockDim.x + threadIdx.x;

    float temp_sum = 0;
    if(row < vec_len)
    {
        for(int col = 0; col < vec_len; col++ )
        {
            temp_sum += B_d[row*vec_len+col] * C_d[col];
        }
        A_d[row] = temp_sum;
    }
}

int main()
{
    int vec_len = 10;
   
    float* C = get_vector(vec_len);
    float** B = get_matrix(vec_len, vec_len);

    float* B_linear = convert_2D_to_1D(B, vec_len, vec_len);

    float *B_d, *C_d, *A_d, *A;
    auto B_size = vec_len * vec_len * sizeof(float);
    auto C_size = vec_len * sizeof(float);
    auto A_size = vec_len * sizeof(float);

    hipMalloc(B_d, B_size);
    hipMalloc(C_d, C_size);

    hipMemcpy(B_d, B,B_size , hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, C_size, hipMemcpyHostToDevice);
    delete[] B;
    delete C;

    dim3 DimGrid(std::ceil(vec_len/2),1,1);
    dim3 DimThread(2,1,1);

    matrix_vec_mult_kernel<<<DimGrid, DimThread>>(A_d, B_d, C_d, vec_len);

    hipMemcpy(A, A_d, A_size, hipMemcpyDeviceToHost);

    hipFree(B_d);
    hipFree(C_d);
    hipFree(A_d);


    return 0;
}