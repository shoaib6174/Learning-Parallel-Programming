#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "my_utils.h"

// nvcc matrix_mult_1block_1dthread.cu my_utils.cpp -o matrix_mult_1block_2dthread -x cu


__global__ void  mat_mult_kernel(int* A_d,int A_row, int A_col, int *B_d, int B_row, int  B_col, int* C_d)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int row = i / B_col;
	int col = i % B_col;

	int temp_sum = 0;
	int row_curr = row * A_col;
	for (int k = 0; k < A_col; k++)
	{
		temp_sum += A_d[row_curr + k] * B_d[k*B_row + col];
	}

	C_d[i] = temp_sum;


}

{
	int * linear = new int[r * c];

	for (int i = 0; i < r; i++)
	{
		for (int j = 0; j <c; j++)
		{
			linear[i*c + j] = matrix[i][j];
		}
	}
	return linear;
}

int main()
{
	int A_row = 5;
	int A_col = 4;


	int B_row = 4;
	int B_col = 4;

	int** A = get_matrix(A_row, A_col);
	int** B = get_matrix(B_row, B_col);


	print_matrix(A, A_row, A_col);
	print_matrix(B, B_row, B_col);

	int* A_linear = convert_2D_to_1D(A, A_row, A_col);
	int* B_linear = convert_2D_to_1D(B, B_row, B_col);

	print_array_as_matrix(A_linear, A_row, A_col);
	int* C = new int[A_row * B_col];

	int *A_d, *B_d, *C_d;
	int A_size = A_row * A_col * sizeof(int);
	int B_size = B_row * B_col * sizeof(int);
	int C_size = A_row * B_col * sizeof(int);

	hipMalloc((void **) &A_d, A_size);
	hipMalloc((void **) &B_d, B_size);
	hipMalloc((void **) &C_d, C_size);

	hipMemcpy(A_d, A_linear, A_size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_linear, B_size, hipMemcpyHostToDevice);

	mat_mult_kernel<<<1,A_row*B_col>>>(A_d, A_row, A_col, B_d, B_row, B_col, C_d);
	hipDeviceSynchronize();

	hipMemcpy(C, C_d,C_size, hipMemcpyDeviceToHost);

	print_array_as_matrix(C, A_row, B_col);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	free(A);
	free(B);
	free(C);
	return 0;
}
