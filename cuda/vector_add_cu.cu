#include <iostream>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>
#include <stdio.h>

void print_vector(int* vec, int n)
{
	for(int i = 0; i < n-1; i++)
	{
		std::cout << vec[i] << " , ";	
	}
	std::cout << vec[n-1] << "\n";
}

__global__ void add_vectors_kernel(int* A_d, int* B_d, int* C_d, int n)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
	{
		C_d[i] = A_d[i] + B_d[i];	
	}
}


int* get_vector(int size)
{
	int* vec = new int[size];
	std::random_device rd;
	std::mt19937 gen(rd());	
	std::uniform_int_distribution<> distr(0, 100);

	for (int i = 0; i < size; i++)
	{
		vec[i] = distr(gen);
	}
	return vec;
}

int main()
{
	int n  = 10;
	int size = n * sizeof(int);
	int* A = get_vector(n);
	int* B = get_vector(n);
	int* C = new int[n];

	print_vector(A, n);
	print_vector(B, n);

	int *A_d,*B_d, *C_d;

	hipMalloc((void **) &A_d, size);
	hipMalloc((void **) &B_d, size);
	hipMalloc((void **) &C_d, size);

	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);


	int num_of_threads = 256;
	dim3 DimGrid((n+num_of_threads) / num_of_threads, 1, 1);
	dim3 DimThread(num_of_threads, 1, 1);

	add_vectors_kernel<<<DimGrid, DimThread>>>(A_d, B_d, C_d, n);

	hipDeviceSynchronize();
	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

	print_vector(C,n);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	return 0;
}
