#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <random>

void print_vector(int* vec, int size)
{
	for(int i = 0; i < size-1; i++)
	{
		std::cout << vec[i] << " , ";	
	}
	std::cout << vec[size] << "\n";
}

__global__ void add_vectors_kernel(int* A, int* B, int* C, int n)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
	{
		C[i] = A[i] + B[i];	
	}
}


int* get_vector(int size)
{
	int* vec = new int[size];
	std::random_device rd;
	std::mt19937 gen(rd());
	
	std::uniform_int_distribution<> distr(0, 100);
	
	for (int i = 0; i < size; i++)
	{
		vec[i] = distr(gen);
	}	
	return vec;
}

int main(int argc, char* argv[])
{	
	int n  =atoi(argv[1]);
	int size = n * sizeof(int);
	int* A = get_vector(n);
	int* B = get_vector(n);
	int* C = new int[n];

	print_vector(A, n);
	print_vector(B, n);
	
	int *A_d,*B_d, *C_d;

	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &A_d, size);
	hipMalloc((void **) &B_d, size);
	
	int num_of_threads = 256;
	dim3 DimGrid(std::ceil(n / num_of_threads), 1, 1);
	dim3 DimThread(num_of_thread, 1, 1);
	
	add_vectors_kernel<<DimGrid, DimThread>>(A_d, B_d, C_d, n);
	
	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	return 0;
	
}
