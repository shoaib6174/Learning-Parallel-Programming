#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

int**  get_matrix(int row, int col, int v=1)
{
	std::cout << "creating matrix" << "\n";
	

	int** matrix = new int*[row];
	
	for(int i = 0; i < row; i++)
	{
		matrix[i] = new int[col];
	}

	for(int i=0; i < row; i++)
	{
		for(int j = 0; j < col; j++)
		{
			matrix[i][j] = v;
		}
	}
	
		
	return matrix;
}


__global__ void  mat_mult_kernel(int* A_d,int A_row, int A_col, int *B_d, int B_row, int  B_col, int* C_d)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int row = i / B_col;
	int col = i % B_col;
	
	int temp_sum = 0;
	int row_t = row * A_col;

	for (int k = 0; i < A_row; i++)
	{	
		temp_sum += A_d[row_t + k] * B_d[k*B_row + col];	
	}
	
	C_d[row * A_row + col] = temp_sum;
	

}

void print_matrix(int** matrix, int row, int col)
{	
	std::cout << "Printing Matrix: " << "\n";
	for(int i = 0; i < row; i++)
	{
		for(int j = 0; j < col; j++)
		{
			std::cout << matrix[i][j] << " , ";
		}
		std::cout << "\n";
	}
}
void print_array_as_vector(int *A, r, c)
{
	for(int i =0; i < r*c; i++)
	{
		std::cout << A[i] << " , ";
		
		if (i % c == 0)
		{
			std::cout << "\n";
		}
	}
}

int* convert_2D_to_1D(int* matrix_start, r, c)
{
	int * linear = new int[r * c];
	
	for (int i = 0; i < r * c; i++)
	{
		linear[i] = matrix_start[i];
	}
	return linear;
}

int main()
{	
	std::cout << "argc = " << argc << "\n";	
	int A_row = 3;
	int A_col = 3;
	
	
	int B_row = 3;
	int B_col = 5;

	int** A = get_matrix(A_row, A_col);
	int** B = get_matrix(B_row, B_col);


	print_matrix(A, A_row, A_col);	
	print_matrix(B, B_row, B_col);
	
	int* A_linear = convert_2D_to_1D(&A[0][0], A_row, A_col);
	int* B_linear = convert_2D_to_1D(&B[0][0], B_row, B_col);
	
	int* C = new int[A_row * B_col]; 
	
	int *A_d, *B_d, *C_d;
	int A_size = A_row * A_col * sizeof(int);
	int B_size = B_row * B_col * sizeof(int);
	int C_size = A_row * B_col * sizeof(int);
	
	hipMemcpy(A_d, A_linear, A_size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_linear, B_size, hipMemcpyHostToDevice); 
	
	mat_mult_kernel<<<1,A_row*B_col>>>(A_d, A_row, A_col, B_d, B_row, B_col, C_d);
	
	hipMemcpy(C, C_d,C_size, hipMemcpyDeviceToHost);
 
	print_matrix(C, A_row, B_col);
	return 0;	
}
