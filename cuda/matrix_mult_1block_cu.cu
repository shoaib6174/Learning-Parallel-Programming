
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

int**  get_matrix(int row, int col, int v=1)
{
	std::cout << "creating matrix" << "\n";

	int** matrix = new int*[row];

	for(int i = 0; i < row; i++)
	{
		matrix[i] = new int[col];
	}

	for(int i=0; i < row; i++)
	{
		for(int j = 0; j < col; j++)
		{
			matrix[i][j] = v;
		}
	}

	return matrix;
}


__global__ void  mat_mult_kernel(int* A_d,int A_row, int A_col, int *B_d, int B_row, int  B_col, int* C_d)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int row = i / B_col;
	int col = i % B_col;

	int temp_sum = 0;
	int row_curr = row * A_col;
	for (int k = 0; k < A_col; k++)
	{
		temp_sum += A_d[row_curr + k] * B_d[k*B_row + col];
	}

	C_d[i] = temp_sum;


}

void print_matrix(int** matrix, int row, int col)
{
	std::cout << "Printing Matrix: " << "\n";
	for(int i = 0; i < row; i++)
	{
		for(int j = 0; j < col; j++)
		{
			std::cout << matrix[i][j] << " , ";
		}
		std::cout << "\n";
	}
}
void print_array_as_matrix(int *A,int r,int c)
{
	std::cout << "Printing Matrix: \n";
	for(int i =0; i < r*c; i++)
	{
		std::cout << A[i] << " , ";

		if (i>0 & (i+1) % c == 0)
		{
			std::cout << "\n";
		}
	}
	std::cout << "\n";
}

int* convert_2D_to_1D(int** matrix,int r,int c)
{
	int * linear = new int[r * c];

	for (int i = 0; i < r; i++)
	{
		for (int j = 0; j <c; j++)
		{
			linear[i*c + j] = matrix[i][j];
		}
	}
	return linear;
}

int main()
{
	int A_row = 5;
	int A_col = 4;


	int B_row = 4;
	int B_col = 4;

	int** A = get_matrix(A_row, A_col);
	int** B = get_matrix(B_row, B_col);


	print_matrix(A, A_row, A_col);
	print_matrix(B, B_row, B_col);

	int* A_linear = convert_2D_to_1D(A, A_row, A_col);
	int* B_linear = convert_2D_to_1D(B, B_row, B_col);

	print_array_as_matrix(A_linear, A_row, A_col);
	int* C = new int[A_row * B_col];

	int *A_d, *B_d, *C_d;
	int A_size = A_row * A_col * sizeof(int);
	int B_size = B_row * B_col * sizeof(int);
	int C_size = A_row * B_col * sizeof(int);

	hipMalloc((void **) &A_d, A_size);
	hipMalloc((void **) &B_d, B_size);
	hipMalloc((void **) &C_d, C_size);

	hipMemcpy(A_d, A_linear, A_size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_linear, B_size, hipMemcpyHostToDevice);

	mat_mult_kernel<<<1,A_row*B_col>>>(A_d, A_row, A_col, B_d, B_row, B_col, C_d);
	hipDeviceSynchronize();

	hipMemcpy(C, C_d,C_size, hipMemcpyDeviceToHost);

	print_array_as_matrix(C, A_row, B_col);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	free(A);
	free(B);
	free(C);
	return 0;
}
