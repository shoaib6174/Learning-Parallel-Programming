#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "my_utils.h"
#include<cmath>

// nvcc matrix_mult_1block_1dthread.cu my_utils.cpp -o matrix_mult_1block_2dthread -x cu


__global__ void  mat_mult_kernel(int* A_d,int A_row, int A_col, int *B_d, int B_row, int  B_col, int* C_d)
{

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	// int col = blockIdx.y * blockDim.y + threadIdx.y;
    for(int col = 0; col < B_col; col++)
    {
        int temp_sum = 0;
        int row_curr = row * A_col;
        for (int k = 0; k < A_col; k++)
        {
            temp_sum += A_d[row_curr + k] * B_d[k * B_row + col];
        }

        C_d[row * B_col + col ] = temp_sum;
    }
	
}



int main()
{
	int A_row = 5;
	int A_col = 4;


	int B_row = 4;
	int B_col = 4;

	int** A = get_matrix(A_row, A_col);
	int** B = get_matrix(B_row, B_col);


	print_matrix(A, A_row, A_col);
	print_matrix(B, B_row, B_col);

	int* A_linear = convert_2D_to_1D(A, A_row, A_col);
	int* B_linear = convert_2D_to_1D(B, B_row, B_col);

	print_array_as_matrix(A_linear, A_row, A_col);
	int* C = new int[A_row * B_col];

	int *A_d, *B_d, *C_d;
	int A_size = A_row * A_col * sizeof(int);
	int B_size = B_row * B_col * sizeof(int);
	int C_size = A_row * B_col * sizeof(int);

	hipMalloc((void **) &A_d, A_size);
	hipMalloc((void **) &B_d, B_size);
	hipMalloc((void **) &C_d, C_size);

	hipMemcpy(A_d, A_linear, A_size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_linear, B_size, hipMemcpyHostToDevice);

	dim3 DimGrid(1,1,1);
	dim3 DimThread(A_row, 1, 1 );

	mat_mult_kernel<<<DimGrid, DimThread>>>(A_d, A_row, A_col, B_d, B_row, B_col, C_d);
	hipDeviceSynchronize();

	hipMemcpy(C, C_d,C_size, hipMemcpyDeviceToHost);

	print_array_as_matrix(C, A_row, B_col);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	free(A);
	free(B);
	free(C);
	return 0;
}
