#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "my_utils.h"
#include<cmath>
#include <hip/hip_runtime.h>

// nvcc matrix_mult_1block_1dthread.cu my_utils.cpp -o matrix_mult_1block_2dthread -x cu

// Error checking macro
#define CUDA_CHECK_ERROR(call)                                               \
do {                                                                         \
    hipError_t err = call;                                                  \
    if (err != hipSuccess) {                                                \
        std::cerr << "CUDA error in file '" << __FILE__ << "' at line "      \
                  << __LINE__ << ": " << hipGetErrorString(err) << std::endl;\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
} while(0)


__global__ void  mat_mult_kernel(int* A_d,int A_row, int A_col, int *B_d, int B_row, int  B_col, int* C_d)
{

	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	int temp_sum = 0;
	int row_curr = row * A_col;
	for (int k = 0; k < A_col; k++)
	{
		temp_sum += A_d[row_curr + k] * B_d[k * B_row + col];
	}

	C_d[row * B_col + col ] = temp_sum;
}



int main()
{
	int A_row = 5;
	int A_col = 4;


	int B_row = 4;
	int B_col = 4;

	int** A = get_matrix(A_row, A_col);
	int** B = get_matrix(B_row, B_col);


//	print_matrix(A, A_row, A_col);
//	print_matrix(B, B_row, B_col);

	int* A_linear = convert_2D_to_1D(A, A_row, A_col);
	int* B_linear = convert_2D_to_1D(B, B_row, B_col);

//	print_array_as_matrix(A_linear, A_row, A_col);
	int* C = new int[A_row * B_col];

	int *A_d, *B_d, *C_d;
	int A_size = A_row * A_col * sizeof(int);
	int B_size = B_row * B_col * sizeof(int);
	int C_size = A_row * B_col * sizeof(int);

	hipMalloc((void **) &A_d, A_size);
	hipMalloc((void **) &B_d, B_size);
	hipMalloc((void **) &C_d, C_size);

	hipMemcpy(A_d, A_linear, A_size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_linear, B_size, hipMemcpyHostToDevice);

	dim3 DimGrid(std::ceil(A_row / 4),std::ceil(B_col / 4),1);
	dim3 DimThread(4, 4, 1 );

	mat_mult_kernel<<<DimGrid, DimThread>>>(A_d, A_row, A_col, B_d, B_row, B_col, C_d);
	// Check for any errors from the kernel launch
    CUDA_CHECK_ERROR(hipGetLastError());

    // Synchronize the device and check for any runtime errors
    CUDA_CHECK_ERROR(hipDeviceSynchronize());

	hipMemcpy(C, C_d,C_size, hipMemcpyDeviceToHost);

	print_array_as_matrix(C, A_row, B_col);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
	free(A);
	free(B);
	free(C);
	return 0;
}
