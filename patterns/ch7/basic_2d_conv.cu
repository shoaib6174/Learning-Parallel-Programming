#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include "my_utils.h"
#include <cmath>

#define thread_size 2;

// Error checking macro
#define CUDA_CHECK_ERROR(call)                                               \
do {                                                                         \
    hipError_t err = call;                                                  \
    if (err != hipSuccess) {                                                \
        std::cerr << "CUDA error in file '" << __FILE__ << "' at line "      \
                  << __LINE__ << ": " << hipGetErrorString(err) << std::endl;\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
} while(0)

__global__
void conv2d_kernel(float* I, float* F, float* R, int r, int row, int col)
{   
    int f_size = 2*r+1;
    
    outRow = blockIdx.x * blockDim.x + threadIdx.x;
    outCol = blockIdx.y * blockDim.y + threadIdx.y;

    float temp_sum = 0.0;
    for (int fRow = 0 ; fRow < f_size; fRow++)
    {
        for(int fCol = 0 ; fCol < f_size; fCol++)
        {   
            int inRow = outRow - r + fRow;
            int inCol = outCol - r + fCol;
            if(inRow>=0 && inRow < row && inCol >= 0 && inCol < col)
            {
                temp_sum += F[fRow * f_size + fCol] * I[inRow * row + inCol]  ;
            }
        }
    }
    
    R[outRow*row+outCol] = temp_sum;

}

void conv2d(float* I, float* F, float* R, int r, int row, int col)
{   
    int f_size = 2*r+1;
    for(int outRow = 0; outRow < row; outRow++)
    {
        for(int outCol=0;  outCol < col; outCol++)
        {
            float temp_sum = 0.0;
            for (int fRow = 0 ; fRow < f_size; fRow++)
            {
                for(int fCol = 0 ; fCol < f_size; fCol++)
                {   
                    int inRow = outRow - r + fRow;
                    int inCol = outCol - r + fCol;
                    if(inRow>=0 && inRow < row && inCol >= 0 && inCol < col)
                    {
                        temp_sum += F[fRow * f_size + fCol] * I[inRow * row + inCol]  ;
                    }
                }
            }
            
            R[outRow*row+outCol] = temp_sum;
        }
    }

}

int main(int argc, char* argv[])
{   
    int row = std::atoi(argv[1]);
    int col = std::atoi(argv[2]);
    float v = std::atof(argv[3]);

    int r = std::atoi(argv[4]);
    float* filter = get_matrix_1d(2* r + 1, 2* r + 1 , 1.0 / pow(2* r + 1, 2) );

    float* matrix = get_matrix_1d(row,col, (float) v);

    print_array_as_matrix(matrix, row, col);
    print_array_as_matrix(filter, 2* r + 1, 2* r + 1);
    float* output = new float[row*col]; 
    conv2d(matrix, filter, output, r, row, col);
    print_array_as_matrix(output, row, col);

    // cuda
    int input_size = row * col * sizeof(float);
    int filter_size = pow(2* r + 1, 2) * sizeof(float);

    float *I_d, *F_d, *R_d, *R;

    CUDA_CHECK_ERROR( hipMalloc(I_d, input_size));
    CUDA_CHECK_ERROR( hipMalloc(R_d, input_size));
    CUDA_CHECK_ERROR( hipMalloc(F_d, filter_size));


    CUDA_CHECK_ERROR( hipMemcpy(I_d, matrix, input_size, hipMemcpyHostToDevice) );
    CUDA_CHECK_ERROR( hipMemcpy(F_d, filter, input_size, hipMemcpyHostToDevice) );

    dim3 DimGrid( std::ceil((float) row / thread_size ) , std::ceil((float) col / thread_size), 1 );
    dim3 DimThread(thread_size, thread_size, 1); 

    convKernel <<<DimGrid, DimThread>>>(I_d, F_d, R_d, r, row, col);
  
    CUDA_CHECK_ERROR(hipGetLastError());
	CUDA_CHECK_ERROR(hipDeviceSynchronize());

    CUDA_CHECK_ERROR( hipMemcpy(R, R_d, input_size, hipMemcpyDeviceToHost) );
    print_array_as_matrix(R, row, col);

    return 0;
}